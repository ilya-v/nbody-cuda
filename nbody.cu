
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdbool.h>
#include <ctype.h>

static clock_t t0;

void startTimer() {
    t0 = clock();
}

double getTimer() {
    return (clock() - t0) / (double)CLOCKS_PER_SEC;
}

#define BLOCK_SIZE 256

typedef struct {

    unsigned
        n_steps,
        n_steps_for_report,
        n_steps_for_output;

    double
        dt_max,
        dv_max,
        dt_start,
        t_start,
        r2_eps;
} params_t;


static params_t params = {
    .n_steps = 1000*1000,
    .n_steps_for_report = 100,
    .n_steps_for_output = 1000,
    .dt_max = 0.01,
    .dv_max = 0.01,
    .dt_start = 0.001,
    .t_start = 0,
    .r2_eps = 0.01
};

typedef struct  {
    const char *name, *type;
    void *ptr;
} param_rec_t;

static const param_rec_t param_recs[] = {
    {   "n_steps",              "%u",   &params.n_steps             },
    {   "n_steps_for_report",   "%u",   &params.n_steps_for_report  },
    {   "n_steps_for_output",   "%u",   &params.n_steps_for_output  },
    {   "dt_max",               "%lf",  &params.dt_max              },
    {   "dv_max",               "%lf",  &params.dv_max              },
    {   "dt_start",             "%lf",  &params.dt_start            },
    {   "t_start",              "%lf",  &params.t_start             },
    {   "r2_eps",               "%lf",  &params.r2_eps              },
    {   NULL,   }
};

bool try_read_param(const char *line, const param_rec_t *rec) {
    const char *key = strstr(line, rec->name);
    const char *key_end = key? (key + strlen(rec->name)) : NULL;
    const bool key_found = key_end &&
        (*key_end == '=' || isspace(*key_end));

    const char *value = key_found? strchr(key, '=') : NULL;
    return  value?  (sscanf(value + 1, rec->type, rec->ptr) == 1) : false;
}

void read_params() {

    FILE *fparam = fopen("params.txt", "r");
    if (!fparam)
        return;

    const unsigned max_line_len = 256;
    for(char buf[max_line_len] = {'\x0',}; fgets(buf, max_line_len, fparam);)
        for (const param_rec_t *rec = param_recs; rec->name; rec++)
            if (try_read_param(buf, rec))
                break;

    fclose(fparam);
}

void show_params(const bool to_stdout) {
    FILE *fo = to_stdout? stdout : stderr;
    for (const param_rec_t *rec = param_recs; rec->name; rec++) {
        (0 == strcmp("%lf", rec->type))?
            fprintf(fo, "#%s = %lg\n", rec->name, *(double*)rec->ptr) :
        (0 == strcmp("%u", rec->type))?
            fprintf(fo, "#%s = %u\n", rec->name, *(unsigned*)rec->ptr) :
            fprintf(fo, "#%s = unknown\n", rec->name);
    }
}

typedef struct  {
    unsigned step;
    double t, dt, ep, ek, etot, I;
} status_t;
status_t status = { 0, };

typedef struct {
    const char *fmt;
    void *ptr;
} status_rec_t;

static const status_rec_t status_recs[] = {
    { "step %8u",        &status.step    },
    { "time %16.8lf",    &status.t       },
    { "dt   %16.8le",    &status.dt      },
    { "ep   %16.8le",    &status.ep      },
    { "ek   %16.8le",    &status.ek      },
    { "etot %16.8le",    &status.etot    },
    { "I    %16.8le",    &status.I       },
    { NULL  }
};

void print_double(const char *fmt, void *p) { printf(fmt, *(double*)p); }
void print_int   (const char *fmt, void *p) { printf(fmt, *(int*)p);    }
typedef struct {
    const char *type;
    void (*print_f)(const char *fmt, void *p);
} print_rec_t;
static const print_rec_t print_recs[] = {
    { "lf", print_double    },
    { "lg", print_double    },
    { "le", print_double    },
    { "u",  print_int       },
    { "d",  print_int       },
    { NULL  }
};

void status_print_header() {
    for (const status_rec_t *rec = status_recs; rec->fmt; rec++) {
        const int
            width = atoi(strchr(rec->fmt, '%') + 1),
            n = (int)(strchr(rec->fmt, ' ') - rec->fmt);
        printf("%*s#%*.*s", width - n, " ", n, n, rec->fmt );
    }
    printf("\n");
}

void status_print() {
    for (const status_rec_t *rec = status_recs; rec->fmt; rec++) {
        char *type = NULL;
        strtod(strchr(rec->fmt, '%') + 1, &type);
        for (const print_rec_t * prec = print_recs; prec->type; prec++) {
            if(strncmp(type, prec->type, strlen(prec->type)) == 0) {
                prec->print_f(strchr(rec->fmt, '%') - 1, rec->ptr);
                break;
            }
        }
    }
    printf("\n");
}


typedef struct { double x, y, z, vx, vy, vz; } Particle;


void status_update( const unsigned N,
                    const Particle particles[],
                    const double u[])
{
    double px = 0, py = 0, pz = 0;
    status.ek = 0;
    status.I = 0;
    for (const Particle *p = particles; p < particles + N; p++) {
        px += p->vx;
        py += p->vy;
        pz += p->vz;
        status.ek += (p->vx*p->vx + p->vy*p->vy + p->vz*p->vz)/2;
        status.I += p->x*p->x + p->y*p->y + p->z*p->z;
    }

    status.ep = 0;
    for (unsigned i = 0; i < N; i++)
        status.ep += u[i];

    status.etot = status.ek - status.ep;
}

void particles_center(const unsigned N, Particle particles[]) {

    double  x= 0,   y = 0,  z = 0;
    double  px = 0, py = 0, pz = 0;
    for (Particle *p = particles; p < particles + N; p++) {
        px += p->vx;    py += p->vy;    pz += p->vz;
        x += p->x;      y += p->y;      z += p->z;
    }
    px/=N;  py/=N;  pz/=N;
    x/=N;   y/=N;   z/=N;
    for (Particle *p = particles; p < particles + N; p++) {
        p->vx -= px;    p->vy -= py;    p->vz -= pz;
        p->x -= x;      p->y -= y;      p->z -= z;
    }
}

__global__ void calcForces(Particle *p, double dt, unsigned N, double r2_eps) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        double fx = 0, fy = 0, fz = 0;
        for (unsigned j = 0; j < N; j++) {
            const double
                dx = p[j].x - p[i].x,
                dy = p[j].y - p[i].y,
                dz = p[j].z - p[i].z,
                distSqr = dx*dx + dy*dy + dz*dz + r2_eps,
                invDist = rsqrt(distSqr),
                invDist3 = invDist * invDist * invDist;

            fx += dx * invDist3;
            fy += dy * invDist3;
            fz += dz * invDist3;
        }

        p[i].vx += dt*fx;
        p[i].vy += dt*fy;
        p[i].vz += dt*fz;
    }
}

__device__ double d_potential = 0;

__global__
void calcPotential(Particle *p, double *u, unsigned N, double r2_eps) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        double ui = 0;
        for (unsigned j = i + 1; j < N; j++) {
            const double
                dx = p[j].x - p[i].x,
                dy = p[j].y - p[i].y,
                dz = p[j].z - p[i].z,
                distSqr = dx*dx + dy*dy + dz*dz + r2_eps;
            ui += rsqrt(distSqr);
        }
        u[i] = ui;
    }
}


int main(const int argc, const char** argv) {

    if (argc >= 2 && strstr(argv[1], "-p"))
    {
        show_params(true);
        return 0;
    }

    read_params();
    show_params(false);

    Particle
        *particles = NULL,
        *old_particles = NULL;

    unsigned N = 0;

    {
        FILE *fin = fopen("input.txt", "r");
        if (!fin) {
            printf("Cannot open input.txt\n");
            return -1;
        }

        {
            double t1, t2, t3, t4, t5, t6;
            for (; 6 == fscanf(fin, "%lf  %lf  %lf  %lf  %lf  %lf\n",
                               &t1, &t2, &t3, &t4, &t5, &t6);
                   N++);
        }

        rewind(fin);
        particles = (Particle *) malloc(N*sizeof(Particle));
        old_particles = (Particle *) malloc(N*sizeof(Particle));
        unsigned i = 0;
        for (; i < N; i++)
            if (6 != fscanf(fin, "%lf  %lf  %lf  %lf  %lf  %lf\n",
                                    &particles[i].x,
                                    &particles[i].y,
                                    &particles[i].z,
                                    &particles[i].vx,
                                    &particles[i].vy,
                                    &particles[i].vz))
                break;

        fclose(fin);

        if (i < N) {
            printf("Cannot read input.txt: %u lines from %u\n", i, N);
            return -1;
        }
    }

    particles_center(N, particles);

    const int nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    Particle *d_p;
    hipMalloc(&d_p, N*sizeof(Particle));

    double *d_u;
    hipMalloc(&d_u, N*sizeof(double));

    status_print_header();

    status.t = params.t_start;
    status.dt = params.dt_start;
    startTimer();
    for (status.step = 0; status.step < params.n_steps; status.step++) {

        hipMemcpy(d_p, particles, N*sizeof(Particle), hipMemcpyHostToDevice);
        calcForces<<<nBlocks, BLOCK_SIZE>>>(d_p, status.dt, N, params.r2_eps);
        Particle *tmp_particles = old_particles;
        old_particles = particles;
        particles = tmp_particles;
        hipMemcpy(particles, d_p, N*sizeof(Particle), hipMemcpyDeviceToHost);

        double dv = 0;
        for (int i = 0 ; i < N; i++) {
            Particle *p = particles + i;
            p->x += p->vx*status.dt;
            p->y += p->vy*status.dt;
            p->z += p->vz*status.dt;

            Particle *op = old_particles + i;
            const double
                dv_ix = p->vx - op->vx,
                dv_iy = p->vy - op->vy,
                dv_iz = p->vz - op->vz,
                dv_i = sqrt(dv_ix * dv_ix + dv_iy*dv_iy + dv_iz*dv_iz);
            if (dv_i > dv)
                dv = dv_i;
        }
        status.t += status.dt;

        status.dt = params.dv_max/dv * status.dt;
        if (status.dt > params.dt_max)
            status.dt = params.dt_max;

        if (status.step % params.n_steps_for_report == 0) {

            static double *u = (double *) malloc(N * sizeof(double));
            for (unsigned i = 0; i < N; i++) u[i] = 0;
            hipMemcpy(d_u, u, N*sizeof(double), hipMemcpyHostToDevice);
            calcPotential<<<nBlocks, BLOCK_SIZE>>>(d_p, d_u, N, params.r2_eps);
            hipMemcpy(u, d_u, N*sizeof(double), hipMemcpyDeviceToHost);

            status_update(N, particles, u);
            status_print();
        }

        if (status.step % params.n_steps_for_output == 0) {
            char fname[256];
            sprintf(fname, "out-%06u.txt", status.step);
            FILE *fout = fopen(fname, "w");
            for (unsigned i = 0; i < N; i++) {
                Particle *p = particles + i;
                fprintf(fout, "%lf %lf %lf %lf %lf %lf\n",
                        p->x, p->y, p->z, p->vx, p->vy, p->vz);
            }

            fclose(fout);
        }
    }

    printf("#N=%d, Steps=%u Titer=%0.3lf s\n",
        N, params.n_steps, getTimer() / params.n_steps);
    free(particles);
    hipFree(d_p);
    hipFree(d_u);
}
