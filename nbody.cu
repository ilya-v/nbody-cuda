
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

static clock_t t0;

void startTimer() {
    t0 = clock();
}

double getTimer() {
    clock_t t = clock();
    return (t - t0) / (double)CLOCKS_PER_SEC;
}

#define BLOCK_SIZE 256
#define SOFTENING 1e-3f

typedef struct { double x, y, z, vx, vy, vz; } Particle;

__global__ void calcForces(Particle *p, double dt, unsigned N) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        double fx = 0, fy = 0, fz = 0;
        for (unsigned j = 0; j < N; j++) {
            const double
                dx = p[j].x - p[i].x,
                dy = p[j].y - p[i].y,
                dz = p[j].z - p[i].z,
                distSqr = dx*dx + dy*dy + dz*dz + SOFTENING,
                invDist = rsqrt(distSqr),
                invDist3 = invDist * invDist * invDist;

            fx += dx * invDist3;
            fy += dy * invDist3;
            fz += dz * invDist3;
        }

        p[i].vx += dt*fx;
        p[i].vy += dt*fy;
        p[i].vz += dt*fz;
    }
}

__device__ double d_potential = 0;

__global__ void calcPotential(Particle *p, double *u, unsigned N) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        double ui = 0;
        for (unsigned j = i + 1; j < N; j++) {
            const double
                dx = p[j].x - p[i].x,
                dy = p[j].y - p[i].y,
                dz = p[j].z - p[i].z,
                distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            ui += rsqrt(distSqr);
        }
        u[i] = ui;
    }
}


int main(const int argc, const char** argv) {

    const int
        nSteps = 10000,
        nStepsForReport = 10;

    const double
        dt = 0.0001f;

    Particle *particles = NULL;
    double *u = NULL;
    unsigned N = 0;

    {
        FILE *fin = fopen("input.txt", "rb");
        if (!fin) {
            printf("Cannot open input.txt\n");
            return -1;
        }

        {
            double t1, t2, t3, t4, t5, t6;
            for (; 6 == fscanf(fin, "%lf  %lf  %lf  %lf  %lf  %lf\n",
                               &t1, &t2, &t3, &t4, &t5, &t6);
                   N++);
        }

        rewind(fin);
        particles = (Particle *) malloc(N * sizeof(Particle));
        unsigned i = 0;
        for (; i < N; i++)
            if (6 != fscanf(fin, "%lf  %lf  %lf  %lf  %lf  %lf\n",
                                    &particles[i].x,
                                    &particles[i].y,
                                    &particles[i].z,
                                    &particles[i].vx,
                                    &particles[i].vy,
                                    &particles[i].vz))
                break;

        fclose(fin);

        if (i < N) {
            printf("Cannot read input.txt: %u lines from %u\n", i, N);
            return -1;
        }
        u = (double *) malloc(N * sizeof(double));
    }

    const int nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    Particle *d_p;
    hipMalloc(&d_p, N*sizeof(Particle));

    double *d_u;
    hipMalloc(&d_u, N*sizeof(double));

    startTimer();
    for (unsigned step = 0; step < nSteps; step++) {

        hipMemcpy(d_p, particles, N*sizeof(Particle), hipMemcpyHostToDevice);
        calcForces<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, N);
        hipMemcpy(particles, d_p, N*sizeof(Particle), hipMemcpyDeviceToHost);

        for (int i = 0 ; i < N; i++) {
            particles[i].x += particles[i].vx*dt;
            particles[i].y += particles[i].vy*dt;
            particles[i].z += particles[i].vz*dt;
        }

        if (step % nStepsForReport == 0) {
            double px = 0, py = 0, pz = 0;
            double ek = 0;
            for (int i = 0 ; i < N; i++) {
                Particle *p = particles + i;
                px += p->vx;
                py += p->vy;
                pz += p->vz;
                ek += (p->vx*p->vx + p->vy*p->vy + p->vz*p->vz)/2;
            }


            for (unsigned i = 0; i < N; i++)
                u[i] = 0;
            hipMemcpy(d_u, u, N*sizeof(double), hipMemcpyHostToDevice);
            calcPotential<<<nBlocks, BLOCK_SIZE>>>(d_p, d_u, N);
            hipMemcpy(u, d_u, N*sizeof(double), hipMemcpyDeviceToHost);
            double ep = 0;
            for (unsigned i = 0; i < N; i++)
                ep += u[i];

            printf("i %u t %lf p %lf %lf %lf Ep %lf Ek %lf E %lf\n",
                step, getTimer(), px, py, pz, ep, ek, ek + ep);
        }
    }

    printf("N=%d, Steps=%u Titer=%0.3lf s\n", N, nSteps, getTimer() / nSteps);
    free(particles);
    hipFree(d_p);
    hipFree(d_u);
}
